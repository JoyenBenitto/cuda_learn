#include "hip/hip_runtime.h"


#include <stdlib.h>
#include <stdio.h>

//Device Code
__global__ void unique_idx_calc_threadIdx(int * input) {
	int tid = threadIdx.x;
	printf("threadIdx.x : %d, value : %d \n", tid, input[tid]);
}

//Host code
int main() {

    int array_size = 8;
    int array_byte_size = sizeof(int) * array_size;
    int h_data[] = {31, 34, 41, 44, 23, 32, 34, 23};

	for(int i = 0; i < array_size; i++){
		printf("%d ", h_data[i]);
	}
	printf("\n \n");

	int * d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(8);
	dim3 grid(1);
	unique_idx_calc_threadIdx <<<grid, block>>>(d_data);
	
    hipDeviceSynchronize();
    
    hipDeviceReset();
    return 0;
}