#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>

//Device Code
__global__ void hello_cuda() {
    printf("Hello from CUDA world \n");
}

//Host code
int main() {


    //kernel launch parameters

    dim3 block(4);
    dim3 grid(8);

    hello_cuda << <grid, block >> > (); // async call
    printf("Hello from CPU \n");
    hipDeviceSynchronize();// will make the prgram stall till all the launched kernels have finished execution


    hipDeviceReset();
    return 0;
}
