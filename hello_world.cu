#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>

//Device Code
__global__ void hello_cuda(){
    printf("Hello from CUDA world \n");
}

//Host code
int main(){
    

    //kernel launch parameters
    hello_cuda<< <1,1>> > (); // async call
    printf("Hello from CPU \n");
    hipDeviceSynchronize();// will make the prgram stall till all the launched kernels have finished execution
    

    hipDeviceReset();
    return 0;
}